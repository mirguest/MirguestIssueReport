#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Template project which demonstrates the basics on how to setup a project
* example application.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>
// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

#define CONSTANT_LS_BALL_R 17500.
#define CONSTANT_LS_RINDEX 1.50
#define CONSTANT_WATER_RINDEX 1.33
#define CONSTANT_PMT_BALL_R 19500.
#define CONSTANT_MEAN_PATH_LS 60000.
#define CONSTANT_RAYLEIGH_LS 30000.
#define CONSTANT_MEAN_PATH_WATER 30000.


////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);

extern "C"
void computeGold(float *reference, float *idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void
testKernel(float *g_idata, float *g_odata)
{
    // shared memory
    // the size is determined by the host application
    extern  __shared__  float sdata[];

    // access thread id
    const unsigned int tid = threadIdx.x;
    // access number of threads in this block
    const unsigned int num_threads = blockDim.x;

    // read in input data from global memory
    sdata[tid] = g_idata[tid];
    __syncthreads();

    // perform some computations
    sdata[tid] = (float) num_threads * sdata[tid];
    __syncthreads();

    // write data to global memory
    g_odata[tid] = sdata[tid];
}

__global__ void
init_rand_state(hiprandState* state, unsigned int seed) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    hiprand_init(seed, id, 0, &state[id]);
}

__global__ void
generate_op_uniform(hiprandState *state,
                    float* op_px, float* op_py, float* op_pz,
                    float* op_polx, float* op_poly, float* op_polz) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    /* Copy state to local memory for efficiency */
    hiprandState localState = state[id];

    // == theta and phi ==
    float costheta = -1. + 2*hiprand_uniform(&localState);
    float sintheta = sqrtf(1-costheta*costheta);
    float phi = 2*HIP_PI_F*hiprand_uniform(&localState);
    float cosphi = cosf(phi);
    float sinphi = sinf(phi);

    op_px[id] = 1.*sintheta*cosf(phi);
    op_py[id] = 1.*sintheta*sinf(phi);
    op_pz[id] = 1.*costheta;

    // == in a local coordinate, generate polarization in x-y plane ==
    float pol_phi = 2*HIP_PI_F*hiprand_uniform(&localState);
    float dx = cosf(pol_phi);
    float dy = sinf(pol_phi);
    // === rotate the polarization ===
    op_polx[id] = cosphi*costheta*dx - sinphi*dy;
    op_poly[id] = sinphi*costheta*dx + cosphi*dy;
    op_polz[id] = -sintheta*dx;;

    /* Copy state back to global memory */
    state[id] = localState;
}

__device__ void
rotateUz(const float& u1, const float& u2, const float& u3, 
         float& dx, float& dy, float& dz) {
    // rotate (dx, dy, dz) to (dx', dy', dz')
    // copy from CLHEP::ThreeVector::rotateUz
    float up = u1*u1 + u2*u2;

    if (up>0) {
        up = sqrtf(up);
        double px = dx,  py = dy,  pz = dz;
        dx = (u1*u3*px - u2*py)/up + u1*pz;
        dy = (u2*u3*px + u1*py)/up + u2*pz;
        dz =    -up*px +             u3*pz;
    } else if (u3 < 0.) {
        dx = -dx;
        dz = -dz;
    }
}

__device__ void
do_rayleigh(hiprandState& state,
            float& op_px,   float& op_py,   float& op_pz,
            float& op_polx, float& op_poly, float& op_polz) {
    float sc_op_px, sc_op_py, sc_op_pz;
    float sc_op_polx, sc_op_poly, sc_op_polz;

    int cnt = 0;
    while(true) {
        // == sample the scattering momentum ==
        // === sample the scattering momentum in local coordiniate ===
        float costheta = -1. + 2*hiprand_uniform(&state);
        float sintheta = sqrtf(1-costheta*costheta);
        float phi = 2*HIP_PI_F*hiprand_uniform(&state);
        float cosphi = cosf(phi);
        float sinphi = sinf(phi);

        sc_op_px = 1.*sintheta*cosphi;
        sc_op_py = 1.*sintheta*sinphi;
        sc_op_pz = 1.*costheta;
        // === rotate the scattering momentum in global coordiniate ===
        rotateUz(op_px, op_py, op_pz, sc_op_px, sc_op_py, sc_op_pz);

        // == caculate the scattering polarization ==
        // pol_sc = (pol - cos(alpha) n)/sin(alpha)
        // alpha is the angle between n and pol.
        float cosalpha = op_polx * sc_op_px
                       + op_poly * sc_op_py
                       + op_polz * sc_op_pz;
        float sinalpha = sqrtf(1.-cosalpha*cosalpha);
        sc_op_polx = (op_polx - cosalpha*sc_op_px)/sinalpha;
        sc_op_poly = (op_poly - cosalpha*sc_op_py)/sinalpha;
        sc_op_polz = (op_polz - cosalpha*sc_op_pz)/sinalpha;

        // == sample using cos(theta)**2 ==
        // === cos(theta_pol) = pol dot sc_pol ===
        float costhetap = op_polx*sc_op_polx
                        + op_poly*sc_op_poly
                        + op_polz*sc_op_polz;
        if (hiprand_uniform(&state) <= powf(costhetap,2)) {
            break;
        }
        // FIXME
        if (++cnt>100) {
            break;
        }
    }

    op_px = sc_op_px;
    op_py = sc_op_py;
    op_pz = sc_op_pz;

    op_polx = sc_op_polx;
    op_poly = sc_op_poly;
    op_polz = sc_op_polz;
}

__device__ int 
stepping_in_LS(hiprandState& localState,
        float& op_x,    float& op_y,    float& op_z,   float& op_t,
        float& op_px,   float& op_py,   float& op_pz,
        float& op_polx, float& op_poly, float& op_polz
        ) {

    // == do the sampling of length ==
    // === calculate the absorption length ===
    float dist_abs = CONSTANT_MEAN_PATH_LS * (-logf(hiprand_uniform(&localState)));
    // === calculate the rayleigh scattering length ===
    float dist_ray = CONSTANT_RAYLEIGH_LS*(-logf(hiprand_uniform(&localState)));
    // === calculate the propagation length to boundary ===
    float r2 = ( op_x*op_x + op_y*op_y + op_z*op_z);
    float r_costheta = (op_x*op_px + op_y*op_py + op_z*op_pz); 
    float r_sintheta = sqrtf(r2 - r_costheta*r_costheta);
    float dist = - r_costheta + sqrtf(CONSTANT_LS_BALL_R + r_sintheta)
                               *sqrtf(CONSTANT_LS_BALL_R - r_sintheta);
    // === select the minimal dist ===
    // type of physics
    // * 0 -> flight
    // * 1 -> absorption
    // * 2 -> rayleigh
    int type = 0;

    if (dist_abs <= dist) {
        // the photon stop at the abs position
        dist = dist_abs;
        type = 1;
    }
    if (dist_ray <= dist) {
        // the photon stop at the rayleigh position
        dist = dist_ray;
        type = 2;
    }

    // == update the position and time ==
    op_x += dist*op_px;
    op_y += dist*op_py;
    op_z += dist*op_pz;
    // TODO calculate the time

    // == update the momentum and polarization ==
    if (type == 1) {
        // the photon stop at the abs position
        op_px = 0.0;
        op_py = 0.0;
        op_pz = 0.0;
    } else if (type == 2) {
        // update momentum
        do_rayleigh(localState, op_px,   op_py,   op_pz,
                                op_polx, op_poly, op_polz);
    }

    return type;
}

__global__ void 
propagate_op_to_boundary(hiprandState *state,
                    float* op_x,  float* op_y,  float* op_z, float* op_t,
                    float* op_px, float* op_py, float* op_pz,
                    float* op_polx, float* op_poly, float* op_polz
                    ) {
    float dist = -1.0; // if dist < 0, some errors happen

    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (op_px[id] == 0.0 && op_py[id] == 0.0 && op_pz[id] == 0.0) {
        return;
    }
    /* Copy state to local memory for efficiency */
    hiprandState localState = state[id];

    while (true) {
        int type = stepping_in_LS(localState,
                    op_x[id],    op_y[id],    op_z[id],   op_t[id],
                    op_px[id],   op_py[id],   op_pz[id],
                    op_polx[id], op_poly[id], op_polz[id]
                    );
        if (type == 0 || type == 1) {
            break;
        }
    }


    /* Copy state back to global memory */
    state[id] = localState;
}

__global__ void 
propagate_op_at_boundary(hiprandState *state,
                    float* op_x,  float* op_y,  float* op_z,
                    float* op_px, float* op_py, float* op_pz) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (op_px[id] == 0.0 && op_py[id] == 0.0 && op_pz[id] == 0.0) {
        return;
    }

    float n = CONSTANT_LS_RINDEX / CONSTANT_WATER_RINDEX;
    // incident: 
    float r2 = ( op_x[id]*op_x[id] + op_y[id]*op_y[id] + op_z[id]*op_z[id]);
    float r = sqrtf(r2);
    float norm_x = op_x[id]/r;
    float norm_y = op_y[id]/r;
    float norm_z = op_z[id]/r;
    // r \dot dir = r * cos(theta)
    float cosI = (op_x[id]*op_px[id]
                + op_y[id]*op_py[id]
                + op_z[id]*op_pz[id]
                  ) / r; 
    float sinT2 = n*n*(1.0-cosI*cosI);
    if (sinT2 > 1.0) {
        // total internal reflection
        // * for current stage, just set it to zero.
        op_px[id] = 0.0;
        op_py[id] = 0.0;
        op_pz[id] = 0.0;
    } else {
        // refraction
        float tmp = (sqrtf(1.0 - sinT2)-n*cosI);

        op_px[id] = n*op_px[id] + tmp * norm_x;
        op_py[id] = n*op_py[id] + tmp * norm_y;
        op_pz[id] = n*op_pz[id] + tmp * norm_z;
    }



}

__global__ void 
propagate_op_to_boundary_pmt(hiprandState *state,
                    float* op_x,  float* op_y,  float* op_z,
                    float* op_px, float* op_py, float* op_pz) {
    float dist = -1.0; // if dist < 0, some errors happen

    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (op_px[id] == 0.0 && op_py[id] == 0.0 && op_pz[id] == 0.0) {
        return;
    }
    /* Copy state to local memory for efficiency */
    hiprandState localState = state[id];

    float r2 = ( op_x[id]*op_x[id] + op_y[id]*op_y[id] + op_z[id]*op_z[id]);
    float r = sqrtf(r2);
    // r \dot dir = r * cos(theta)
    float r_costheta = (op_x[id]*op_px[id]
                      + op_y[id]*op_py[id]
                      + op_z[id]*op_pz[id]
                        ); 
    
    dist = - r_costheta + sqrtf( CONSTANT_PMT_BALL_R*CONSTANT_PMT_BALL_R
                               - (r2 - r_costheta*r_costheta));
    // == absorption ==
    float dist_abs = CONSTANT_MEAN_PATH_WATER * (-logf(hiprand_uniform(&localState)));
    if (dist_abs <= dist) {
        // the photon stop at the abs position
        dist = dist_abs;
    }
    // update the position
    op_x[id] += dist*op_px[id];
    op_y[id] += dist*op_py[id];
    op_z[id] += dist*op_pz[id];
    if (dist_abs <= dist) {
        // the photon stop at the abs position
        op_px[id] = 0.0;
        op_py[id] = 0.0;
        op_pz[id] = 0.0;
    }
    /* Copy state back to global memory */
    state[id] = localState;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char **argv)
{
    runTest(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest(int argc, char **argv)
{
    bool bTestResult = true;

    printf("%s Starting...\n\n", argv[0]);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    int devID = findCudaDevice(argc, (const char **)argv);

    StopWatchInterface *timer = 0;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    // unsigned int num_threads = 1; //32;
    // unsigned int num_blocks = 1; //64;
    unsigned int num_threads = 32;
    unsigned int num_blocks = 64;

    unsigned int total_photon = num_threads * num_blocks;

    int init_pos_x = 0;
    int init_pos_y = 0;
    int init_pos_z = 0;
    
    if (checkCmdLineFlag(argc, (const char **)argv, "total")) {
        total_photon = getCmdLineArgumentInt(argc, (const char **)argv, "total");
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "x")) {
        init_pos_x = getCmdLineArgumentInt(argc, (const char **)argv, "x");
    }
    if (checkCmdLineFlag(argc, (const char **)argv, "y")) {
        init_pos_y = getCmdLineArgumentInt(argc, (const char **)argv, "y");
    }
    if (checkCmdLineFlag(argc, (const char **)argv, "z")) {
        init_pos_z = getCmdLineArgumentInt(argc, (const char **)argv, "z");
    }


    num_blocks = (total_photon+num_threads-1)/num_threads;


    unsigned int mem_size = sizeof(float) * num_threads;

    // allocate host memory
    float *h_idata = (float *) malloc(mem_size);

    // initalize the memory
    for (unsigned int i = 0; i < num_threads; ++i)
    {
        h_idata[i] = (float) i;
    }

    // allocate device memory
    float *d_idata;
    checkCudaErrors(hipMalloc((void **) &d_idata, mem_size));
    // copy host memory to device
    checkCudaErrors(hipMemcpy(d_idata, h_idata, mem_size,
                               hipMemcpyHostToDevice));

    // allocate device memory for result
    float *d_odata;
    checkCudaErrors(hipMalloc((void **) &d_odata, mem_size));

    // setup execution parameters
    dim3  grid(num_blocks, 1, 1);
    dim3  threads(num_threads, 1, 1);

    // execute the kernel
    //testKernel<<< grid, threads, mem_size >>>(d_idata, d_odata);

    // = OP Tracking =
    // == initialize random generator =
    hiprandState *devStates = 0;
    hipMalloc((void **)&devStates, grid.x * threads.x * 
                              sizeof(hiprandState));

    // == momentum ==
    float *h_oppx = 0;
    float *h_oppy = 0;
    float *h_oppz = 0;
    h_oppx = (float*)malloc(grid.x * threads.x * sizeof(float));
    h_oppy = (float*)malloc(grid.x * threads.x * sizeof(float));
    h_oppz = (float*)malloc(grid.x * threads.x * sizeof(float));

    float *d_oppx = 0;
    float *d_oppy = 0;
    float *d_oppz = 0;

    hipMalloc((void**)&d_oppx, grid.x * threads.x * sizeof(float));
    hipMalloc((void**)&d_oppy, grid.x * threads.x * sizeof(float));
    hipMalloc((void**)&d_oppz, grid.x * threads.x * sizeof(float));

    // == polarization ==
    float *h_oppolx = 0;
    float *h_oppoly = 0;
    float *h_oppolz = 0;
    h_oppolx = (float*)malloc(grid.x * threads.x * sizeof(float));
    h_oppoly = (float*)malloc(grid.x * threads.x * sizeof(float));
    h_oppolz = (float*)malloc(grid.x * threads.x * sizeof(float));

    float *d_oppolx = 0;
    float *d_oppoly = 0;
    float *d_oppolz = 0;

    hipMalloc((void**)&d_oppolx, grid.x * threads.x * sizeof(float));
    hipMalloc((void**)&d_oppoly, grid.x * threads.x * sizeof(float));
    hipMalloc((void**)&d_oppolz, grid.x * threads.x * sizeof(float));

    // == position ==
    // set the initial position
    // default unit is mm, (same as geant4)
    float *h_opx = 0;
    float *h_opy = 0;
    float *h_opz = 0;
    float *h_opt = 0;
    h_opx = (float*)malloc(grid.x * threads.x * sizeof(float));
    h_opy = (float*)malloc(grid.x * threads.x * sizeof(float));
    h_opz = (float*)malloc(grid.x * threads.x * sizeof(float));
    h_opt = (float*)malloc(grid.x * threads.x * sizeof(float));

    for (int i = 0; i < grid.x * threads.x; ++i) {
        h_opx[i] = init_pos_x; // 1m
        h_opy[i] = init_pos_y; // 0m
        h_opz[i] = init_pos_z; // 0m
        h_opt[i] = 0.0; // 0m
    }

    float *d_opx = 0;
    float *d_opy = 0;
    float *d_opz = 0;
    float *d_opt = 0;

    hipMalloc((void**)&d_opx, grid.x * threads.x * sizeof(float));
    hipMalloc((void**)&d_opy, grid.x * threads.x * sizeof(float));
    hipMalloc((void**)&d_opz, grid.x * threads.x * sizeof(float));
    hipMalloc((void**)&d_opt, grid.x * threads.x * sizeof(float));

    hipMemcpy(d_opx, h_opx, grid.x * threads.x * sizeof(float),
                        hipMemcpyHostToDevice); 
    hipMemcpy(d_opy, h_opy, grid.x * threads.x * sizeof(float),
                        hipMemcpyHostToDevice); 
    hipMemcpy(d_opz, h_opz, grid.x * threads.x * sizeof(float),
                        hipMemcpyHostToDevice); 
    hipMemcpy(d_opt, h_opt, grid.x * threads.x * sizeof(float),
                        hipMemcpyHostToDevice); 

    // == start ==
    // === initialize the random engine ===
    init_rand_state<<< grid, threads >>>(devStates, 42);
    // === generate the direction ===
    generate_op_uniform<<< grid, threads >>>(devStates, 
            d_oppx, d_oppy, d_oppz,
            d_oppolx, d_oppoly, d_oppolz
            );
    // === start tracking optical photon ===
    // ==== -> LS boundary ====
    // the position will be updated
    propagate_op_to_boundary<<< grid, threads >>>(devStates, 
            d_opx,  d_opy,  d_opz, d_opt,
            d_oppx, d_oppy, d_oppz,
            d_oppolx, d_oppoly, d_oppolz
            );
    // // ==== -> Refract between LS and Water====
    // // the momentum will be updated
    // propagate_op_at_boundary<<< grid, threads >>>(devStates, 
    //         d_opx,  d_opy,  d_opz,
    //         d_oppx, d_oppy, d_oppz);
    // // ==== -> PMT boundary ====
    // // the position will be updated
    // propagate_op_to_boundary_pmt<<< grid, threads >>>(devStates, 
    //         d_opx,  d_opy,  d_opz,
    //         d_oppx, d_oppy, d_oppz);

    // check if kernel execution generated and error
    getLastCudaError("Kernel execution failed");

    // == copy data back to host ==
    // === copy data back to host (position)===
    hipMemcpy(h_opx, d_opx, grid.x * threads.x * sizeof(float),
                        hipMemcpyDeviceToHost);
    hipMemcpy(h_opy, d_opy, grid.x * threads.x * sizeof(float),
                        hipMemcpyDeviceToHost);
    hipMemcpy(h_opz, d_opz, grid.x * threads.x * sizeof(float),
                        hipMemcpyDeviceToHost);
    // === copy data back to host (momentum)===
    hipMemcpy(h_oppx, d_oppx, grid.x * threads.x * sizeof(float),
                        hipMemcpyDeviceToHost);
    hipMemcpy(h_oppy, d_oppy, grid.x * threads.x * sizeof(float),
                        hipMemcpyDeviceToHost);
    hipMemcpy(h_oppz, d_oppz, grid.x * threads.x * sizeof(float),
                        hipMemcpyDeviceToHost);


    // allocate mem for the result on host side
    float *h_odata = (float *) malloc(mem_size);
    // copy result from device to host
    checkCudaErrors(hipMemcpy(h_odata, d_odata, sizeof(float) * num_threads,
                               hipMemcpyDeviceToHost));

    sdkStopTimer(&timer);
    printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);


    // = print the data =
    // == print the data of momentum ==
    for (int i = 0; i < grid.x*threads.x; ++i) {
        std::cout << h_oppx[i] << " " << h_oppy[i] << " " << h_oppz[i] << std::endl;
    }
    std::cout << "========================================================"
              << std::endl;
    // == print the data of position ==
    for (int i = 0; i < grid.x*threads.x; ++i) {
        std::cout << h_opx[i] << " " << h_opy[i] << " " << h_opz[i] << std::endl;
    }

    // compute reference solution
    // float *reference = (float *) malloc(mem_size);
    // computeGold(reference, h_idata, num_threads);

    // check result
    // if (checkCmdLineFlag(argc, (const char **) argv, "regression"))
    // {
    //     // write file for regression test
    //     sdkWriteFile("./data/regression.dat", h_odata, num_threads, 0.0f, false);
    // }
    // else
    // {
    //     // custom output handling when no regression test running
    //     // in this case check if the result is equivalent to the expected soluion
    //     bTestResult = compareData(reference, h_odata, num_threads, 0.0f, 0.0f);
    // }

    // cleanup memory
    free(h_idata);
    free(h_odata);
    // free(reference);
    checkCudaErrors(hipFree(d_idata));
    checkCudaErrors(hipFree(d_odata));

    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits   
    hipDeviceReset();
    exit(bTestResult ? EXIT_SUCCESS : EXIT_FAILURE);
}
