#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Template project which demonstrates the basics on how to setup a project
* example application.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>
// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);

extern "C"
void computeGold(float *reference, float *idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void
testKernel(float *g_idata, float *g_odata)
{
    // shared memory
    // the size is determined by the host application
    extern  __shared__  float sdata[];

    // access thread id
    const unsigned int tid = threadIdx.x;
    // access number of threads in this block
    const unsigned int num_threads = blockDim.x;

    // read in input data from global memory
    sdata[tid] = g_idata[tid];
    __syncthreads();

    // perform some computations
    sdata[tid] = (float) num_threads * sdata[tid];
    __syncthreads();

    // write data to global memory
    g_odata[tid] = sdata[tid];
}

__global__ void
init_rand_state(hiprandState* state, unsigned int seed) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    hiprand_init(seed, id, 0, &state[id]);
}

__global__ void
generate_op_uniform(hiprandState *state,
                    float* op_px, float* op_py, float* op_pz) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    /* Copy state to local memory for efficiency */
    hiprandState localState = state[id];

    // == theta and phi ==
    float costheta = -1. + 2*hiprand_uniform(&localState);
    float sintheta = sqrtf(1-costheta*costheta);
    float phi = 2*HIP_PI_F*hiprand_uniform(&localState);

    op_px[id] = 1.*sintheta*cosf(phi);
    op_py[id] = 1.*sintheta*sinf(phi);
    op_pz[id] = 1.*costheta;

    /* Copy state back to global memory */
    state[id] = localState;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char **argv)
{
    runTest(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest(int argc, char **argv)
{
    bool bTestResult = true;

    printf("%s Starting...\n\n", argv[0]);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    int devID = findCudaDevice(argc, (const char **)argv);

    StopWatchInterface *timer = 0;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    unsigned int num_threads = 32;
    unsigned int num_blocks = 64;
    unsigned int mem_size = sizeof(float) * num_threads;

    // allocate host memory
    float *h_idata = (float *) malloc(mem_size);

    // initalize the memory
    for (unsigned int i = 0; i < num_threads; ++i)
    {
        h_idata[i] = (float) i;
    }

    // allocate device memory
    float *d_idata;
    checkCudaErrors(hipMalloc((void **) &d_idata, mem_size));
    // copy host memory to device
    checkCudaErrors(hipMemcpy(d_idata, h_idata, mem_size,
                               hipMemcpyHostToDevice));

    // allocate device memory for result
    float *d_odata;
    checkCudaErrors(hipMalloc((void **) &d_odata, mem_size));

    // setup execution parameters
    dim3  grid(num_blocks, 1, 1);
    dim3  threads(num_threads, 1, 1);

    // execute the kernel
    //testKernel<<< grid, threads, mem_size >>>(d_idata, d_odata);

    // random generator
    hiprandState *devStates = 0;
    hipMalloc((void **)&devStates, grid.x * threads.x * 
                              sizeof(hiprandState));

    float *h_oppx = 0;
    float *h_oppy = 0;
    float *h_oppz = 0;
    h_oppx = (float*)malloc(grid.x * threads.x * sizeof(float));
    h_oppy = (float*)malloc(grid.x * threads.x * sizeof(float));
    h_oppz = (float*)malloc(grid.x * threads.x * sizeof(float));

    float *d_oppx = 0;
    float *d_oppy = 0;
    float *d_oppz = 0;

    hipMalloc((void**)&d_oppx, grid.x * threads.x * sizeof(float));
    hipMalloc((void**)&d_oppy, grid.x * threads.x * sizeof(float));
    hipMalloc((void**)&d_oppz, grid.x * threads.x * sizeof(float));

    init_rand_state<<< grid, threads >>>(devStates, 42);

    generate_op_uniform<<< grid, threads >>>(devStates, d_oppx, d_oppy, d_oppz);

    // check if kernel execution generated and error
    getLastCudaError("Kernel execution failed");

    // == copy data back to host ==
    hipMemcpy(h_oppx, d_oppx, grid.x * threads.x * sizeof(float),
                        hipMemcpyDeviceToHost);
    hipMemcpy(h_oppy, d_oppy, grid.x * threads.x * sizeof(float),
                        hipMemcpyDeviceToHost);
    hipMemcpy(h_oppz, d_oppz, grid.x * threads.x * sizeof(float),
                        hipMemcpyDeviceToHost);


    // allocate mem for the result on host side
    float *h_odata = (float *) malloc(mem_size);
    // copy result from device to host
    checkCudaErrors(hipMemcpy(h_odata, d_odata, sizeof(float) * num_threads,
                               hipMemcpyDeviceToHost));

    sdkStopTimer(&timer);
    printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);


    // print the data
    for (int i = 0; i < grid.x*threads.x; ++i) {
        std::cout << h_oppx[i] << " " << h_oppy[i] << " " << h_oppz[i] << std::endl;
    }

    // compute reference solution
    // float *reference = (float *) malloc(mem_size);
    // computeGold(reference, h_idata, num_threads);

    // check result
    // if (checkCmdLineFlag(argc, (const char **) argv, "regression"))
    // {
    //     // write file for regression test
    //     sdkWriteFile("./data/regression.dat", h_odata, num_threads, 0.0f, false);
    // }
    // else
    // {
    //     // custom output handling when no regression test running
    //     // in this case check if the result is equivalent to the expected soluion
    //     bTestResult = compareData(reference, h_odata, num_threads, 0.0f, 0.0f);
    // }

    // cleanup memory
    free(h_idata);
    free(h_odata);
    // free(reference);
    checkCudaErrors(hipFree(d_idata));
    checkCudaErrors(hipFree(d_odata));

    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits   
    hipDeviceReset();
    exit(bTestResult ? EXIT_SUCCESS : EXIT_FAILURE);
}
