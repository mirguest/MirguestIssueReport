#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>
// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

#define CONSTANT_LS_BALL_R 17500.
#define CONSTANT_LS_RINDEX 1.50
#define CONSTANT_WATER_RINDEX 1.33
#define CONSTANT_PMT_BALL_R 19500.
#define CONSTANT_MEAN_PATH_LS 60000.
#define CONSTANT_RAYLEIGH_LS 30000.
#define CONSTANT_MEAN_PATH_WATER 30000.

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);

////////////////////////////////////////////////////////////////////////////////
__global__ void
init_rand_state(hiprandState* state, unsigned int seed) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    hiprand_init(seed, id, 0, &state[id]);
}

////////////////////////////////////////////////////////////////////////////////
__global__ void
generate_op_uniform(hiprandState *state,
                    float* op_px,   float* op_py,   float* op_pz,
                    float* op_polx, float* op_poly, float* op_polz) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    /* Copy state to local memory for efficiency */
    hiprandState localState = state[id];

    // == theta and phi ==
    float costheta = -1. + 2*hiprand_uniform(&localState);
    float sintheta = sqrtf(1-costheta*costheta);
    float phi = 2*HIP_PI_F*hiprand_uniform(&localState);
    float cosphi = cosf(phi);
    float sinphi = sinf(phi);

    op_px[id] = 1.*sintheta*cosphi;
    op_py[id] = 1.*sintheta*sinphi;
    op_pz[id] = 1.*costheta;

    // == in a local coordinate, generate polarization in x-y plane ==
    float pol_phi = 2*HIP_PI_F*hiprand_uniform(&localState);
    float dx = cosf(pol_phi);
    float dy = sinf(pol_phi);
    // === rotate the polarization ===
    op_polx[id] = cosphi*costheta*dx - sinphi*dy;
    op_poly[id] = sinphi*costheta*dx + cosphi*dy;
    op_polz[id] = -sintheta*dx;;

    /* Copy state back to global memory */
    state[id] = localState;
}

__device__ void
rotateUz(const float& u1, const float& u2, const float& u3, 
         float& dx, float& dy, float& dz) {
    // rotate (dx, dy, dz) to (dx', dy', dz')
    // copy from CLHEP::ThreeVector::rotateUz
    float up = u1*u1 + u2*u2;

    if (up>0) {
        up = sqrtf(up);
        double px = dx,  py = dy,  pz = dz;
        dx = (u1*u3*px - u2*py)/up + u1*pz;
        dy = (u2*u3*px + u1*py)/up + u2*pz;
        dz =    -up*px +             u3*pz;
    } else if (u3 < 0.) {
        dx = -dx;
        dz = -dz;
    }
}

__device__ void
do_rayleigh(hiprandState& state,
            float& op_px,   float& op_py,   float& op_pz,
            float& op_polx, float& op_poly, float& op_polz) {
    float sc_op_px, sc_op_py, sc_op_pz;
    float sc_op_polx, sc_op_poly, sc_op_polz;

    int cnt = 0;
    while(true) {
        // == sample the scattering momentum ==
        // === sample the scattering momentum in local coordiniate ===
        float costheta = -1. + 2*hiprand_uniform(&state);
        float sintheta = sqrtf(1-costheta*costheta);
        float phi = 2*HIP_PI_F*hiprand_uniform(&state);
        float cosphi = cosf(phi);
        float sinphi = sinf(phi);

        sc_op_px = 1.*sintheta*cosphi;
        sc_op_py = 1.*sintheta*sinphi;
        sc_op_pz = 1.*costheta;
        // === rotate the scattering momentum in global coordiniate ===
        rotateUz(op_px, op_py, op_pz, sc_op_px, sc_op_py, sc_op_pz);

        // == caculate the scattering polarization ==
        // pol_sc = (pol - cos(alpha) n)/sin(alpha)
        // alpha is the angle between n and pol.
        float cosalpha = op_polx * sc_op_px
                       + op_poly * sc_op_py
                       + op_polz * sc_op_pz;
        float sinalpha = sqrtf(1.-cosalpha*cosalpha);
        sc_op_polx = (op_polx - cosalpha*sc_op_px)/sinalpha;
        sc_op_poly = (op_poly - cosalpha*sc_op_py)/sinalpha;
        sc_op_polz = (op_polz - cosalpha*sc_op_pz)/sinalpha;

        // == sample using cos(theta)**2 ==
        // === cos(theta_pol) = pol dot sc_pol ===
        float costhetap = op_polx*sc_op_polx
                        + op_poly*sc_op_poly
                        + op_polz*sc_op_polz;
        if (hiprand_uniform(&state) <= powf(costhetap,2)) {
            break;
        }
        // FIXME
        if (++cnt>100) {
            break;
        }
    }

    op_px = sc_op_px;
    op_py = sc_op_py;
    op_pz = sc_op_pz;

    op_polx = sc_op_polx;
    op_poly = sc_op_poly;
    op_polz = sc_op_polz;
}
////////////////////////////////////////////////////////////////////////////////
__global__ void
propagate_op_rayleigh(hiprandState *state,
                    float* op_px,   float* op_py,   float* op_pz,
                    float* op_polx, float* op_poly, float* op_polz) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    // Copy state to local memory for efficiency 
    hiprandState localState = state[id];

    do_rayleigh(localState, op_px[id],   op_py[id],   op_pz[id],
                            op_polx[id], op_poly[id], op_polz[id]);

    // Copy state back to global memory
    state[id] = localState;
}
////////////////////////////////////////////////////////////////////////////////

int
main(int argc, char **argv)
{
    runTest(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest(int argc, char **argv)
{
    printf("%s Starting...\n\n", argv[0]);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    int devID = findCudaDevice(argc, (const char **)argv);

    StopWatchInterface *timer = 0;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    // unsigned int num_threads = 1; //32;
    // unsigned int num_blocks = 1; //64;
    unsigned int num_threads = 32;
    unsigned int num_blocks = 64;

    unsigned int total_photon = num_threads * num_blocks;

    int init_pos_x = 0;
    int init_pos_y = 0;
    int init_pos_z = 0;

    // setup execution parameters
    dim3  grid(num_blocks, 1, 1);
    dim3  threads(num_threads, 1, 1);

    // == initialize random generator =
    hiprandState *devStates = 0;
    hipMalloc((void **)&devStates, grid.x * threads.x * 
                              sizeof(hiprandState));

    // == position ==
    // set the initial position
    // default unit is mm, (same as geant4)
    float *h_op_x = 0;
    float *h_op_y = 0;
    float *h_op_z = 0;
    h_op_x = (float*)malloc(grid.x * threads.x * sizeof(float));
    h_op_y = (float*)malloc(grid.x * threads.x * sizeof(float));
    h_op_z = (float*)malloc(grid.x * threads.x * sizeof(float));

    for (int i = 0; i < grid.x * threads.x; ++i) {
        h_op_x[i] = init_pos_x; // 0m
        h_op_y[i] = init_pos_y; // 0m
        h_op_y[i] = init_pos_z; // 0m
    }

    float *d_op_x = 0;
    float *d_op_y = 0;
    float *d_op_z = 0;

    hipMalloc((void**)&d_op_x, grid.x * threads.x * sizeof(float));
    hipMalloc((void**)&d_op_y, grid.x * threads.x * sizeof(float));
    hipMalloc((void**)&d_op_z, grid.x * threads.x * sizeof(float));

    hipMemcpy(d_op_x, h_op_x, grid.x * threads.x * sizeof(float),
                        hipMemcpyHostToDevice); 
    hipMemcpy(d_op_y, h_op_y, grid.x * threads.x * sizeof(float),
                        hipMemcpyHostToDevice); 
    hipMemcpy(d_op_z, h_op_z, grid.x * threads.x * sizeof(float),
                        hipMemcpyHostToDevice); 

    // = initialize =
    // == momentum ==
    float *h_op_px = 0;
    float *h_op_py = 0;
    float *h_op_pz = 0;
    h_op_px = (float*)malloc(grid.x * threads.x * sizeof(float));
    h_op_py = (float*)malloc(grid.x * threads.x * sizeof(float));
    h_op_pz = (float*)malloc(grid.x * threads.x * sizeof(float));

    float *d_op_px = 0;
    float *d_op_py = 0;
    float *d_op_pz = 0;

    hipMalloc((void**)&d_op_px, grid.x * threads.x * sizeof(float));
    hipMalloc((void**)&d_op_py, grid.x * threads.x * sizeof(float));
    hipMalloc((void**)&d_op_pz, grid.x * threads.x * sizeof(float));

    // == polarization ==
    float *h_op_polx = 0;
    float *h_op_poly = 0;
    float *h_op_polz = 0;
    h_op_polx = (float*)malloc(grid.x * threads.x * sizeof(float));
    h_op_poly = (float*)malloc(grid.x * threads.x * sizeof(float));
    h_op_polz = (float*)malloc(grid.x * threads.x * sizeof(float));

    float *d_op_polx = 0;
    float *d_op_poly = 0;
    float *d_op_polz = 0;

    hipMalloc((void**)&d_op_polx, grid.x * threads.x * sizeof(float));
    hipMalloc((void**)&d_op_poly, grid.x * threads.x * sizeof(float));
    hipMalloc((void**)&d_op_polz, grid.x * threads.x * sizeof(float));

    // = execute =
    // == initialize the random engine ==
    init_rand_state<<< grid, threads >>>(devStates, 42);
    // == generate optical photons ==
    // === generate the direction ===
    generate_op_uniform<<< grid, threads >>>(devStates, 
            d_op_px,   d_op_py,   d_op_pz,
            d_op_polx, d_op_poly, d_op_polz);

    propagate_op_rayleigh<<< grid, threads >>>(devStates, 
            d_op_px,   d_op_py,   d_op_pz,
            d_op_polx, d_op_poly, d_op_polz);

    // = finalize =
    // == copy data back to host ==
    // === copy data back to host (position)===
    hipMemcpy(h_op_x, d_op_x, grid.x * threads.x * sizeof(float),
                        hipMemcpyDeviceToHost);
    hipMemcpy(h_op_y, d_op_y, grid.x * threads.x * sizeof(float),
                        hipMemcpyDeviceToHost);
    hipMemcpy(h_op_z, d_op_z, grid.x * threads.x * sizeof(float),
                        hipMemcpyDeviceToHost);
    // === copy data back to host (momentum)===
    hipMemcpy(h_op_px, d_op_px, grid.x * threads.x * sizeof(float),
                        hipMemcpyDeviceToHost);
    hipMemcpy(h_op_py, d_op_py, grid.x * threads.x * sizeof(float),
                        hipMemcpyDeviceToHost);
    hipMemcpy(h_op_pz, d_op_pz, grid.x * threads.x * sizeof(float),
                        hipMemcpyDeviceToHost);
    // === copy data back to host (polarization)===
    hipMemcpy(h_op_polx, d_op_polx, grid.x * threads.x * sizeof(float),
                        hipMemcpyDeviceToHost);
    hipMemcpy(h_op_poly, d_op_poly, grid.x * threads.x * sizeof(float),
                        hipMemcpyDeviceToHost);
    hipMemcpy(h_op_polz, d_op_polz, grid.x * threads.x * sizeof(float),
                        hipMemcpyDeviceToHost);

    // = display the results =
    for (int i = 0; i < grid.x*threads.x; ++i) {
        std::cout << h_op_x[i] << " " << h_op_y[i] << " " << h_op_z[i] << " "
                  << h_op_px[i] << " " << h_op_py[i] << " " << h_op_pz[i] << " "
                  << h_op_polx[i] << " " << h_op_poly[i] << " " << h_op_polz[i] << " "
                  << std::endl;
    }

}
