#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>
// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

#define CONSTANT_LS_BALL_R 17500.
#define CONSTANT_LS_RINDEX 1.50
#define CONSTANT_WATER_RINDEX 1.33
#define CONSTANT_PMT_BALL_R 19500.
#define CONSTANT_MEAN_PATH_LS 60000.
#define CONSTANT_RAYLEIGH_LS 30000.
#define CONSTANT_MEAN_PATH_WATER 30000.

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);

////////////////////////////////////////////////////////////////////////////////
__global__ void
init_rand_state(hiprandState* state, unsigned int seed) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    hiprand_init(seed, id, 0, &state[id]);
}

////////////////////////////////////////////////////////////////////////////////
__global__ void
generate_op_uniform(hiprandState *state,
                    float* op_px,   float* op_py,   float* op_pz,
                    float* op_polx, float* op_poly, float* op_polz) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    /* Copy state to local memory for efficiency */
    hiprandState localState = state[id];

    // == theta and phi ==
    float costheta = -1. + 2*hiprand_uniform(&localState);
    float sintheta = sqrtf(1-costheta*costheta);
    float phi = 2*HIP_PI_F*hiprand_uniform(&localState);
    float cosphi = cosf(phi);
    float sinphi = sinf(phi);

    op_px[id] = 1.*sintheta*cosphi;
    op_py[id] = 1.*sintheta*sinphi;
    op_pz[id] = 1.*costheta;

    // == in a local coordinate, generate polarization in x-y plane ==
    float pol_phi = 2*HIP_PI_F*hiprand_uniform(&localState);
    float dx = cosf(pol_phi);
    float dy = sinf(pol_phi);
    // === rotate the polarization ===
    op_polx[id] = cosphi*costheta*dx - sinphi*dy;
    op_poly[id] = sinphi*costheta*dx + cosphi*dy;
    op_polz[id] = -sintheta*dx;;

    /* Copy state back to global memory */
    state[id] = localState;
}
////////////////////////////////////////////////////////////////////////////////

int
main(int argc, char **argv)
{
    runTest(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest(int argc, char **argv)
{
    printf("%s Starting...\n\n", argv[0]);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    int devID = findCudaDevice(argc, (const char **)argv);

    StopWatchInterface *timer = 0;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    // unsigned int num_threads = 1; //32;
    // unsigned int num_blocks = 1; //64;
    unsigned int num_threads = 32;
    unsigned int num_blocks = 64;

    unsigned int total_photon = num_threads * num_blocks;

    int init_pos_x = 0;
    int init_pos_y = 0;
    int init_pos_z = 0;

    // setup execution parameters
    dim3  grid(num_blocks, 1, 1);
    dim3  threads(num_threads, 1, 1);

    // == initialize random generator =
    hiprandState *devStates = 0;
    hipMalloc((void **)&devStates, grid.x * threads.x * 
                              sizeof(hiprandState));

    // == position ==
    // set the initial position
    // default unit is mm, (same as geant4)
    float *h_op_x = 0;
    float *h_op_y = 0;
    float *h_op_z = 0;
    h_op_x = (float*)malloc(grid.x * threads.x * sizeof(float));
    h_op_y = (float*)malloc(grid.x * threads.x * sizeof(float));
    h_op_z = (float*)malloc(grid.x * threads.x * sizeof(float));

    for (int i = 0; i < grid.x * threads.x; ++i) {
        h_op_x[i] = init_pos_x; // 0m
        h_op_y[i] = init_pos_y; // 0m
        h_op_y[i] = init_pos_z; // 0m
    }

    float *d_op_x = 0;
    float *d_op_y = 0;
    float *d_op_z = 0;

    hipMalloc((void**)&d_op_x, grid.x * threads.x * sizeof(float));
    hipMalloc((void**)&d_op_y, grid.x * threads.x * sizeof(float));
    hipMalloc((void**)&d_op_z, grid.x * threads.x * sizeof(float));

    hipMemcpy(d_op_x, h_op_x, grid.x * threads.x * sizeof(float),
                        hipMemcpyHostToDevice); 
    hipMemcpy(d_op_y, h_op_y, grid.x * threads.x * sizeof(float),
                        hipMemcpyHostToDevice); 
    hipMemcpy(d_op_z, h_op_z, grid.x * threads.x * sizeof(float),
                        hipMemcpyHostToDevice); 

    // = initialize =
    // == momentum ==
    float *h_op_px = 0;
    float *h_op_py = 0;
    float *h_op_pz = 0;
    h_op_px = (float*)malloc(grid.x * threads.x * sizeof(float));
    h_op_py = (float*)malloc(grid.x * threads.x * sizeof(float));
    h_op_pz = (float*)malloc(grid.x * threads.x * sizeof(float));

    float *d_op_px = 0;
    float *d_op_py = 0;
    float *d_op_pz = 0;

    hipMalloc((void**)&d_op_px, grid.x * threads.x * sizeof(float));
    hipMalloc((void**)&d_op_py, grid.x * threads.x * sizeof(float));
    hipMalloc((void**)&d_op_pz, grid.x * threads.x * sizeof(float));

    // == polarization ==
    float *h_op_polx = 0;
    float *h_op_poly = 0;
    float *h_op_polz = 0;
    h_op_polx = (float*)malloc(grid.x * threads.x * sizeof(float));
    h_op_poly = (float*)malloc(grid.x * threads.x * sizeof(float));
    h_op_polz = (float*)malloc(grid.x * threads.x * sizeof(float));

    float *d_op_polx = 0;
    float *d_op_poly = 0;
    float *d_op_polz = 0;

    hipMalloc((void**)&d_op_polx, grid.x * threads.x * sizeof(float));
    hipMalloc((void**)&d_op_poly, grid.x * threads.x * sizeof(float));
    hipMalloc((void**)&d_op_polz, grid.x * threads.x * sizeof(float));

    // = execute =
    // == initialize the random engine ==
    init_rand_state<<< grid, threads >>>(devStates, 42);
    // == generate optical photons ==
    // === generate the direction ===
    generate_op_uniform<<< grid, threads >>>(devStates, 
            d_op_px,   d_op_py,   d_op_pz,
            d_op_polx, d_op_poly, d_op_polz);

    // = finalize =
    // == copy data back to host ==
    // === copy data back to host (position)===
    hipMemcpy(h_op_x, d_op_x, grid.x * threads.x * sizeof(float),
                        hipMemcpyDeviceToHost);
    hipMemcpy(h_op_y, d_op_y, grid.x * threads.x * sizeof(float),
                        hipMemcpyDeviceToHost);
    hipMemcpy(h_op_z, d_op_z, grid.x * threads.x * sizeof(float),
                        hipMemcpyDeviceToHost);
    // === copy data back to host (momentum)===
    hipMemcpy(h_op_px, d_op_px, grid.x * threads.x * sizeof(float),
                        hipMemcpyDeviceToHost);
    hipMemcpy(h_op_py, d_op_py, grid.x * threads.x * sizeof(float),
                        hipMemcpyDeviceToHost);
    hipMemcpy(h_op_pz, d_op_pz, grid.x * threads.x * sizeof(float),
                        hipMemcpyDeviceToHost);
    // === copy data back to host (polarization)===
    hipMemcpy(h_op_polx, d_op_polx, grid.x * threads.x * sizeof(float),
                        hipMemcpyDeviceToHost);
    hipMemcpy(h_op_poly, d_op_poly, grid.x * threads.x * sizeof(float),
                        hipMemcpyDeviceToHost);
    hipMemcpy(h_op_polz, d_op_polz, grid.x * threads.x * sizeof(float),
                        hipMemcpyDeviceToHost);

    // = display the results =
    for (int i = 0; i < grid.x*threads.x; ++i) {
        std::cout << h_op_x[i] << " " << h_op_y[i] << " " << h_op_z[i] << " "
                  << h_op_px[i] << " " << h_op_py[i] << " " << h_op_pz[i] << " "
                  << h_op_polx[i] << " " << h_op_poly[i] << " " << h_op_polz[i] << " "
                  << std::endl;
    }

}
